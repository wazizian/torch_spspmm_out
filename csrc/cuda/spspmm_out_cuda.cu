#include "../utils.h"
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <ATen/cuda/HIPContext.h>

template <typename scalar_t>
void spspmm_out_cuda(struct csr_matrix<scalar_t>* A, struct csr_matrix<scalar_t>* B, struct csr_matrix<scalar_t>* C){
  struct csc_matrix<scalar_t> cscB(B->nrow, B->ncol, B->nnz);

  
}
