#include <torch/extension.h>
#include <torch/script.h>
#include "utils.h"

#include "cpu/spspmm_out_cpu.h"

#ifdef __HIPCC__
#include "cuda/spspmm_out_cuda.h"
#endif

#ifdef __CUSP__
#include "cusp/spspmm_out_cusp.h"
#endif

template <typename scalar_t>
void check_and_make_csr_matrix(torch::Tensor rowptr, torch::Tensor col,
                               torch::Tensor val, int ncol, bool is_cuda,
                               struct csr_matrix<scalar_t> * m){
  CHECK_INPUT(rowptr.dim() == 1);
  CHECK_INPUT(col.dim() == 1);
  CHECK_INPUT(val.dim() == 1);
  CHECK_INPUT(col.size(0) == val.size(0));
  CHECK_INT(rowptr);
  CHECK_INT(col);
  // val.scalar_type() == scalar_t
  if (is_cuda) {
    CHECK_CUDA(rowptr);
    CHECK_CUDA(col);
    CHECK_CUDA(val);
  }
  m->nnz = col.size(0);
  m->nrow = rowptr.size(0) - 1;
  m->ncol = ncol;
  m->rowptr = rowptr.data_ptr<int>();
  m->col = col.data_ptr<int>();
  m->val = val.data_ptr<scalar_t>();
  return;
} 

void spspmm_out(torch::Tensor rowptrA, torch::Tensor colA, torch::Tensor valA,
                torch::Tensor rowptrB, torch::Tensor colB, torch::Tensor valB, 
                torch::Tensor rowptrC, torch::Tensor colC, torch::Tensor valC,
                int64_t ncolB) {
  auto scalar_type = valA.scalar_type();
  bool is_cuda = false;
  int ncolA = rowptrB.size(0) - 1;
  int ncolC = ncolB;
  if (rowptrA.device().is_cuda()){
#ifdef __HIPCC__
    is_cuda = true;
#else
    AT_ERROR("Not compiled with CUDA support");
#endif
  }
  AT_DISPATCH_FLOATING_TYPES(scalar_type, "spspmm_out", [&] {
      struct csr_matrix<scalar_t> A, B, C;
      check_and_make_csr_matrix<scalar_t>(rowptrA, colA, valA, ncolA, is_cuda, &A);
      check_and_make_csr_matrix<scalar_t>(rowptrB, colB, valB, ncolB, is_cuda, &B);
      check_and_make_csr_matrix<scalar_t>(rowptrC, colC, valC, ncolC, is_cuda, &C);
#ifdef __CUSP__
      spspmm_out_cusp<scalar_t>(is_cuda, A, B, C);
#else
      if (is_cuda){
        AT_ERROR("TODO");
      } else {
        spspmm_out_cpu<scalar_t>(A, B, C);        
      }
#endif
  });
}

static auto registry = torch::RegisterOperators("torch_spspmm_out::spspmm_out", &spspmm_out);
// Another way would be to bind spspmm_out with pybind
// PYBIND11_MODULE(TORCH_EXTENSION_NAME, m){
//   m.def("spspmm_out", &spspmm_out, "SpSpMM out");
// }
